#include "hip/hip_runtime.h"
/*@ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 **
 **    Copyright (c) 2012
 **        Dave A. May [dave.may@erdw.ethz.ch]
 **        Institute of Geophysics
 **        ETH Zürich
 **        Sonneggstrasse 5
 **        CH-8092 Zürich
 **        Switzerland
 **
 **    project:    pTatin3d
 **    filename:   stokes_operators_tensor.c
 **
 **
 **    pTatin3d is free software: you can redistribute it and/or modify
 **    it under the terms of the GNU General Public License as published
 **    by the Free Software Foundation, either version 3 of the License,
 **    or (at your option) any later version.
 **
 **    pTatin3d is distributed in the hope that it will be useful,
 **    but WITHOUT ANY WARRANTY; without even the implied warranty of
 **    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 **    See the GNU General Public License for more details.
 **
 **    You should have received a copy of the GNU General Public License
 **    along with pTatin3d. If not, see <http://www.gnu.org/licenses/>.
 **
 ** ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ @*/
// -*- indent-tabs-mode:t c-basic-offset:8 -*-

#include <petscfe.h>
#include <ptatin3d.h>
#include <ptatin3d_stokes.h>
#include <dmda_element_q2p1.h>


template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \n",
                file, line, result);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define CUDACHECK(val)       check( (val), #val, __FILE__, __LINE__ )


#define NQP 27			/* Number of quadrature points per element; must equal Q2_NODES_PER_EL_3D (27) */

typedef enum {
	GRAD,
	GRAD_TRANSPOSE
} GradMode;



/*
 * Performs three tensor contractions: y[l,a,b,c] += T[a,k] S[b,j] R[c,i] x[l,k,j,i]
 */
static __device__ void TensorContract(PetscReal const *Rf,PetscReal const *Sf,PetscReal const *Tf,GradMode gmode,PetscReal const x[][NQP],PetscReal y[][NQP])
{
	PetscReal R[3][3],S[3][3],T[3][3];
	PetscReal u[3][NQP],v[3][NQP];
    PetscInt i,j,k,l,kj,ji,a,b,c;

	for (j=0; j<3; j++) {
		for (i=0; i<3; i++) {
			R[i][j] = i<3 ? (gmode == GRAD ? Rf[3*i+j] : Rf[3*j + i]) : 0.;
			S[i][j] = i<3 ? (gmode == GRAD ? Sf[3*i+j] : Sf[3*j + i]) : 0.;
			T[i][j] = i<3 ? (gmode == GRAD ? Tf[3*i+j] : Tf[3*j + i]) : 0.;
		}
	}

	// u[l,k,j,c] = R[c,i] x[l,k,j,i]
    for (i=0; i<3; ++i) {
      for (j=0; j<NQP; ++j) {
          u[i][j] = 0;
      }
    }
	for (l=0; l<3; l++) {
		for (kj=0; kj<9; kj++) {
			for (i=0; i<3; i++) {
				for (c=0; c<3; c++) {
					u[l][kj*3+c] += R[c][i] * x[l][kj*3+i];
				}
			}
		}
	}

	// v[l,k,b,c] = S[b,j] u[l,k,j,c]
    for (i=0; i<3; ++i) {
      for (j=0; j<NQP; ++j) {
        v[i][j] = 0;
      }
    }
	for (l=0; l<3; l++) {
		for (k=0; k<3; k++) {
			for (j=0; j<3; j++) {
				for (c=0; c<3; c++) {
					for (b=0; b<3; b++) {
						v[l][(k*3+b)*3+c] += S[b][j] * u[l][(k*3+j)*3+c];
					}
				}
			}
		}
	}

	// y[l,a,b,c] = T[a,k] v[l,k,b,c]
	for (k=0; k<3; k++) {
		for (l=0; l<3; l++) {
			for (a=0; a<3; a++) {
				for (ji=0; ji<9; ji++) {
					y[l][a*9+ji] += T[a][k] * v[l][k*9+ji];
				}
			}
		}
	}
	//PetscLogFlops(3*NQP*(6+6+6));
	//PetscFunctionReturn(0);
}

static __device__ void JacobianInvert(PetscScalar dx[3][3][NQP],PetscScalar dxdet[NQP])
{
	PetscInt i,j,k;

	for (i=0; i<NQP; i++) {
		PetscScalar a[3][3];
		PetscScalar b0,b3,b6,det,idet;
		for (j=0; j<3; j++) {
			for (k=0; k<3; k++) {
				a[j][k] = dx[j][k][i];
			}
		}
		b0 =  (a[1][1]*a[2][2] - a[2][1]*a[1][2]);
		b3 = -(a[1][0]*a[2][2] - a[2][0]*a[1][2]);
		b6 =  (a[1][0]*a[2][1] - a[2][0]*a[1][1]);
		det = a[0][0]*b0 + a[0][1]*b3 + a[0][2]*b6;
		idet = 1.0 / det;
		dx[0][0][i] =  idet*b0;
		dx[0][1][i] = -idet*(a[0][1]*a[2][2] - a[2][1]*a[0][2]);
		dx[0][2][i] =  idet*(a[0][1]*a[1][2] - a[1][1]*a[0][2]);
		dx[1][0][i] =  idet*b3;
		dx[1][1][i] =  idet*(a[0][0]*a[2][2] - a[2][0]*a[0][2]);
		dx[1][2][i] = -idet*(a[0][0]*a[1][2] - a[1][0]*a[0][2]);
		dx[2][0][i] =  idet*b6;
		dx[2][1][i] = -idet*(a[0][0]*a[2][1] - a[2][0]*a[0][1]);
		dx[2][2][i] =  idet*(a[0][0]*a[1][1] - a[1][0]*a[0][1]);
		dxdet[i] =  det;
	}
	//PetscLogFlops(NQP*NEV*(14 + 1/* division */ + 27));
	//return 0;
}

static __device__ void QuadratureAction(const PetscScalar *gaussdata_eta,
				       PetscScalar const dx[3][3][Q2_NODES_PER_EL_3D],
				       PetscScalar const dxdet[Q2_NODES_PER_EL_3D],
				       PetscReal const w[Q2_NODES_PER_EL_3D],
				       PetscScalar const du[3][3][Q2_NODES_PER_EL_3D],
				       PetscScalar dv[3][3][Q2_NODES_PER_EL_3D])
{
	PetscInt i,l,k;

	for (i=0; i<NQP; i++) {
		PetscScalar Du[6],Dv[6]; /* Symmetric gradient with respect to physical coordinates, xx, yy, zz, xy+yx, xz+zx, yz+zy */

		PetscScalar dux[3][3];
		for (l=0; l<3; l++) { // fields
			for (k=0; k<3; k++) { // directions
				dux[k][l] = du[0][l][i] * dx[k][0][i] + du[1][l][i] * dx[k][1][i] + du[2][l][i] * dx[k][2][i];
			}
		}
		Du[0] = dux[0][0];
		Du[1] = dux[1][1];
		Du[2] = dux[2][2];
		Du[3] = 0.5*(dux[0][1] + dux[1][0]);
		Du[4] = 0.5*(dux[0][2] + dux[2][0]);
		Du[5] = 0.5*(dux[1][2] + dux[2][1]);

		for (k=0; k<6; k++) { /* Stress is coefficient of test function */
			Dv[k] = 2 * gaussdata_eta[i] * Du[k];
		}

		PetscScalar dvx[3][3];
		dvx[0][0] = Dv[0];
		dvx[0][1] = Dv[3];
		dvx[0][2] = Dv[4];
		dvx[1][0] = Dv[3];
		dvx[1][1] = Dv[1];
		dvx[1][2] = Dv[5];
		dvx[2][0] = Dv[4];
		dvx[2][1] = Dv[5];
		dvx[2][2] = Dv[2];

		for (l=0; l<3; l++) { // fields
			for (k=0; k<3; k++) { // directions
				dv[k][l][i] = w[i] * dxdet[i] * (dvx[0][l] * dx[0][k][i] + dvx[1][l] * dx[1][k][i] + dvx[2][l] * dx[2][k][i]);
			}
		}
	}
	//PetscLogFlops(NQP*(5*9+6+6+6*9));
	//return 0;
}

static __global__ void MFStokesWrapper_A11_CUDA_kernel(PetscInt nel,PetscInt nen_u,PetscInt const *elnidx_u,PetscReal const *LA_gcoords,PetscScalar const *ufield,PetscReal const *gaussdata,PetscScalar *Yu_premerge,
                                                          PetscReal const *D,PetscReal const *B,PetscReal const *w)
{
	PetscScalar elu[3][Q2_NODES_PER_EL_3D]={},elx[3][Q2_NODES_PER_EL_3D]={},elv[3][Q2_NODES_PER_EL_3D];
	PetscScalar dx[3][3][NQP],dxdet[NQP],du[3][3][NQP],dv[3][3][NQP];
	PetscInt i,j,k,l;
    PetscInt elidx = blockDim.x * blockIdx.x + threadIdx.x;

    if (elidx >= nel)
      return;

	for (i=0; i<Q2_NODES_PER_EL_3D; i++) {
		PetscInt E = elnidx_u[nen_u*elidx+i];
		for (l=0; l<3; l++) {
			elx[l][i] = LA_gcoords[3*E+l];
			elu[l][i] = ufield[3*E+l];
            elv[l][i] = 0;
		}
	}

	for (i=0; i<3; i++) {
      for (j=0; j<3; j++) {
        for (k=0; k<NQP; k++) {
          dx[i][j][k] = 0;
          du[i][j][k] = 0;
          dv[i][j][k] = 0;
        }
      }
    }

	TensorContract(D,B,B,GRAD,elx,dx[0]);
	TensorContract(B,D,B,GRAD,elx,dx[1]);
	TensorContract(B,B,D,GRAD,elx,dx[2]);

	JacobianInvert(dx,dxdet);

	TensorContract(D,B,B,GRAD,elu,du[0]);
	TensorContract(B,D,B,GRAD,elu,du[1]);
	TensorContract(B,B,D,GRAD,elu,du[2]);

	QuadratureAction(gaussdata + elidx*NQP,dx,dxdet,w,du,dv);

	TensorContract(D,B,B,GRAD_TRANSPOSE,dv[0],elv);
	TensorContract(B,D,B,GRAD_TRANSPOSE,dv[1],elv);
	TensorContract(B,B,D,GRAD_TRANSPOSE,dv[2],elv);

    /* Avoid worries about race conditions by applying the merge to Yu on the host! */
	for (i=0; i<NQP; i++) {
		for (l=0; l<3; l++) {
			(Yu_premerge + 3*NQP*elidx)[3*i+l] = elv[l][i];
		}
	}
}

extern "C" {

#undef __FUNCT__
#define __FUNCT__ "MFStokesWrapper_A11_CUDA"
PetscErrorCode MFStokesWrapper_A11_CUDA(Quadrature volQ,DM dau,PetscScalar ufield[],PetscScalar Yu[])
{
	PetscErrorCode ierr;
	DM cda;
	Vec gcoords;
	const PetscReal *LA_gcoords;
	PetscInt nel,nen_u,e,i,j,k,l,localsize;
	const PetscInt *elnidx_u;
	QPntVolCoefStokes *all_gausspoints;
	const QPntVolCoefStokes *cell_gausspoints;
	PetscReal x1[3],w1[3],B[3][3],D[3][3],w[NQP];

	PetscFunctionBegin;
	ierr = PetscDTGaussQuadrature(3,-1,1,x1,w1);CHKERRQ(ierr);
	for (i=0; i<3; i++) {
		B[i][0] = .5*(PetscSqr(x1[i]) - x1[i]);
		B[i][1] = 1 - PetscSqr(x1[i]);
		B[i][2] = .5*(PetscSqr(x1[i]) + x1[i]);
		D[i][0] = x1[i] - .5;
		D[i][1] = -2*x1[i];
		D[i][2] = x1[i] + .5;
	}
	for (i=0; i<3; i++) {
		for (j=0; j<3; j++) {
			for (k=0; k<3; k++) {
				w[(i*3+j)*3+k] = w1[i] * w1[j] * w1[k];}}}

	/* setup for coords */
	ierr = DMGetCoordinateDM( dau, &cda);CHKERRQ(ierr);
	ierr = DMGetCoordinatesLocal( dau,&gcoords );CHKERRQ(ierr);
	ierr = VecGetArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);
    ierr = VecGetLocalSize(gcoords,&localsize);CHKERRQ(ierr);

	ierr = DMDAGetElements_pTatinQ2P1(dau,&nel,&nen_u,&elnidx_u);CHKERRQ(ierr);

	ierr = VolumeQuadratureGetAllCellData_Stokes(volQ,&all_gausspoints);CHKERRQ(ierr);

    /* Set up CUDA data */
    PetscInt *elnidx_u_cuda;
    ierr = hipMalloc(&elnidx_u_cuda,        nel * nen_u * sizeof(PetscInt));CUDACHECK(ierr);
    ierr = hipMemcpy(elnidx_u_cuda,elnidx_u,nel * nen_u * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);

    PetscScalar *ufield_cuda;
    ierr = hipMalloc(&ufield_cuda,       localsize * sizeof(PetscScalar));CUDACHECK(ierr);
    ierr = hipMemcpy(ufield_cuda,ufield, localsize * sizeof(PetscScalar),hipMemcpyHostToDevice);CUDACHECK(ierr);

    PetscReal *LA_gcoords_cuda;
    ierr = hipMalloc(&LA_gcoords_cuda,           localsize * sizeof(PetscReal));CUDACHECK(ierr);
    ierr = hipMemcpy(LA_gcoords_cuda,LA_gcoords, localsize * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);
    
    PetscReal *gaussdata_cuda,*gaussdata_host;
    ierr = PetscMalloc(nel * NQP * sizeof(PetscReal), &gaussdata_host);CHKERRQ(ierr);
    ierr = hipMalloc(&gaussdata_cuda,nel * NQP * sizeof(PetscReal));CUDACHECK(ierr);
    for (e=0; e<nel; e++) {
      ierr = VolumeQuadratureGetCellData_Stokes(volQ,all_gausspoints,e,(QPntVolCoefStokes**)&cell_gausspoints);CHKERRQ(ierr);
      for (i=0; i<NQP; i++) gaussdata_host[e*NQP + i] = cell_gausspoints[i].eta;
    }
    ierr = hipMemcpy(gaussdata_cuda,gaussdata_host, nel * NQP * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    PetscScalar *Yu_premerge_cuda,*Yu_premerge;
    ierr = PetscMalloc(3 * nel * nen_u * sizeof(PetscScalar),&Yu_premerge);CHKERRQ(ierr);
    ierr = hipMalloc(&Yu_premerge_cuda, 3 * nel * nen_u * sizeof(PetscScalar));CUDACHECK(ierr);

    PetscReal *D_cuda;
    ierr = hipMalloc(&D_cuda,  3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
    ierr = hipMemcpy(D_cuda,D, 3 * 3 * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    PetscReal *B_cuda;
    ierr = hipMalloc(&B_cuda,  3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
    ierr = hipMemcpy(B_cuda,B, 3 * 3 * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    PetscReal *w_cuda;
    ierr = hipMalloc(&w_cuda,  3 * 3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
    ierr = hipMemcpy(w_cuda,w, 3 * 3 * 3 * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* CUDA entry point
     *  - inputs: elnidx_u, LA_gcoords, ufield, gaussdata
     *  - output: Yu
     */
    MFStokesWrapper_A11_CUDA_kernel<<<(nel-1)/128 + 1, 128>>>(nel,nen_u,elnidx_u_cuda,LA_gcoords_cuda,ufield_cuda,gaussdata_cuda,Yu_premerge_cuda, D_cuda, B_cuda, w_cuda);

    PetscLogFlops((nel * 9) * 3*NQP*(6+6+6));           /* 9 tensor contractions per element */
    PetscLogFlops(nel*NQP*(14 + 1/* division */ + 27)); /* 1 Jacobi inversion per element */
    PetscLogFlops((nel * 9) * 3*NQP*(6+6+6));           /* 1 quadrature action per element */

    /* Read back CUDA data */
    ierr = hipMemcpy(Yu_premerge,Yu_premerge_cuda, 3 * nel * NQP * sizeof(PetscScalar),hipMemcpyDeviceToHost);CUDACHECK(ierr);

    for (e=0; e<nel; e++) {
      for (i=0; i<NQP; i++) {
		PetscInt E = elnidx_u[nen_u*e+i];
		for (l=0; l<3; l++) {
			Yu[3*E+l] += (Yu_premerge + 3*NQP*e)[3*i+l];
		}
      }
	}

	ierr = VecRestoreArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);

    /* clean up */
    ierr = hipFree(elnidx_u_cuda);CUDACHECK(ierr);
    ierr = hipFree(ufield_cuda);CUDACHECK(ierr);
    ierr = hipFree(LA_gcoords_cuda);CUDACHECK(ierr);
    ierr = PetscFree(gaussdata_host);CHKERRQ(ierr);
    ierr = hipFree(gaussdata_cuda);CUDACHECK(ierr);
    ierr = PetscFree(Yu_premerge);CHKERRQ(ierr);
    ierr = hipFree(Yu_premerge_cuda);CUDACHECK(ierr);
    ierr = hipFree(D_cuda);CUDACHECK(ierr);
    ierr = hipFree(B_cuda);CUDACHECK(ierr);
    ierr = hipFree(w_cuda);CUDACHECK(ierr);

	PetscFunctionReturn(0);
}

} /* extern C */
