#include "hip/hip_runtime.h"
/*@ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 **
 **    Copyright (c) 2012
 **        Dave A. May [dave.may@erdw.ethz.ch]
 **        Institute of Geophysics
 **        ETH Zürich
 **        Sonneggstrasse 5
 **        CH-8092 Zürich
 **        Switzerland
 **
 **    project:    pTatin3d
 **    filename:   stokes_operators_tensor.c
 **
 **
 **    pTatin3d is free software: you can redistribute it and/or modify
 **    it under the terms of the GNU General Public License as published
 **    by the Free Software Foundation, either version 3 of the License,
 **    or (at your option) any later version.
 **
 **    pTatin3d is distributed in the hope that it will be useful,
 **    but WITHOUT ANY WARRANTY; without even the implied warranty of
 **    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 **    See the GNU General Public License for more details.
 **
 **    You should have received a copy of the GNU General Public License
 **    along with pTatin3d. If not, see <http://www.gnu.org/licenses/>.
 **
 ** ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ @*/
// -*- indent-tabs-mode:t c-basic-offset:8 -*-

#include <petscfe.h>
#include <ptatin3d.h>
#include <ptatin3d_stokes.h>
#include <dmda_element_q2p1.h>
#include <stokes_operators.h>

extern PetscLogEvent MAT_MultMFA11_setup;
extern PetscLogEvent MAT_MultMFA11_copyto;
extern PetscLogEvent MAT_MultMFA11_kernel;
extern PetscLogEvent MAT_MultMFA11_copyfrom;
extern PetscLogEvent MAT_MultMFA11_merge;

typedef struct _p_MFA11CUDA *MFA11CUDA;

struct _p_MFA11CUDA {
  PetscObjectState state;

  PetscScalar *ufield;
  PetscReal   *LA_gcoords;
  PetscReal   *gaussdata;
  PetscInt    *elnidx_u;
  PetscScalar *Yu;
};

/* Constant memory for D and B matrices */
__constant__ PetscReal CUDA_D[3*3], CUDA_B[3*3], CUDA_w[3*3*3];


template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d \n",
                file, line, result);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}
#define CUDACHECK(val)       check( (val), #val, __FILE__, __LINE__ )


#define NQP 27			/* Number of quadrature points per element; must equal Q2_NODES_PER_EL_3D (27) */

#define WARPS_PER_BLOCK    4


__device__ double atomicAdd_double(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}


/*
 * Performs three tensor contractions: y[l,a,b,c] += T[a,k] S[b,j] R[c,i] x[l,k,j,i]
 */
static __device__ void TensorContract(PetscReal const *R,PetscReal const *S,PetscReal const *T,PetscReal const x[],PetscReal y[])
{
  __shared__ PetscReal u[WARPS_PER_BLOCK][NQP],v[WARPS_PER_BLOCK][NQP];

  PetscInt warp_in_block = threadIdx.x / 32;
  PetscInt id_in_warp = threadIdx.x % 32;

  PetscInt c = id_in_warp % 3;
  PetscInt kj = id_in_warp / 3;
  PetscInt k3 = (id_in_warp / 9) * 3;
  PetscInt ji = id_in_warp % 9;

  for (PetscInt l=0; l<3; l++) {

	// u[l,k,j,c] = R[c,i] x[l,k,j,i]
    PetscReal result = 0;
    v[warp_in_block][id_in_warp] = x[l];
    for (PetscInt i=0; i<3; i++) result += R[i] * v[warp_in_block][kj*3+i];
    u[warp_in_block][id_in_warp] = result;

	// v[l,k,b,c] = S[b,j] u[l,k,j,c]
    result = 0;
    for (PetscInt j=0; j<3; j++) result += S[j] * u[warp_in_block][(k3+j)*3+c];
    v[warp_in_block][id_in_warp] = result;

	// y[l,a,b,c] = T[a,k] v[l,k,b,c]
    result = 0;
	for (PetscInt k=0; k<3; k++) result += T[k] * v[warp_in_block][k*9+ji];
    y[l] += result;

  } // for l
}

static __device__ void JacobianInvert(PetscScalar dx[3][3],PetscScalar &dxdet)
{
		PetscScalar a[3][3];
		PetscScalar b0,b3,b6,idet;
		for (PetscInt j=0; j<3; j++) {
			for (PetscInt k=0; k<3; k++) {
				a[j][k] = dx[j][k];
			}
		}
		b0 =  (a[1][1]*a[2][2] - a[2][1]*a[1][2]);
		b3 = -(a[1][0]*a[2][2] - a[2][0]*a[1][2]);
		b6 =  (a[1][0]*a[2][1] - a[2][0]*a[1][1]);
		dxdet = a[0][0]*b0 + a[0][1]*b3 + a[0][2]*b6;
		idet = 1.0 / dxdet;
		dx[0][0] =  idet*b0;
		dx[0][1] = -idet*(a[0][1]*a[2][2] - a[2][1]*a[0][2]);
		dx[0][2] =  idet*(a[0][1]*a[1][2] - a[1][1]*a[0][2]);
		dx[1][0] =  idet*b3;
		dx[1][1] =  idet*(a[0][0]*a[2][2] - a[2][0]*a[0][2]);
		dx[1][2] = -idet*(a[0][0]*a[1][2] - a[1][0]*a[0][2]);
		dx[2][0] =  idet*b6;
		dx[2][1] = -idet*(a[0][0]*a[2][1] - a[2][0]*a[0][1]);
		dx[2][2] =  idet*(a[0][0]*a[1][1] - a[1][0]*a[0][1]);
}

static __device__ void QuadratureAction(PetscScalar gaussdata_eta,
				       PetscScalar const dx[3][3],
				       PetscScalar dxdet,
				       PetscReal w,
				       PetscScalar const du[3][3],
				       PetscScalar dv[3][3])
{
		/* Symmetric gradient with respect to physical coordinates, xx, yy, zz, xy+yx, xz+zx, yz+zy */

		PetscScalar dux[3][3];
		for (PetscInt l=0; l<3; l++) { // fields
			for (PetscInt k=0; k<3; k++) { // directions
				dux[k][l] = du[0][l] * dx[k][0] + du[1][l] * dx[k][1] + du[2][l] * dx[k][2];
			}
		}

		PetscScalar dvx[3][3];
		dvx[0][0] = 2 * gaussdata_eta * dux[0][0];
		dvx[0][1] =     gaussdata_eta * (dux[0][1] + dux[1][0]);
		dvx[0][2] =     gaussdata_eta * (dux[0][2] + dux[2][0]);
		dvx[1][0] =     gaussdata_eta * (dux[0][1] + dux[1][0]);
		dvx[1][1] = 2 * gaussdata_eta * dux[1][1];
		dvx[1][2] =     gaussdata_eta * (dux[1][2] + dux[2][1]);
		dvx[2][0] =     gaussdata_eta * (dux[0][2] + dux[2][0]);
		dvx[2][1] =     gaussdata_eta * (dux[1][2] + dux[2][1]);
		dvx[2][2] = 2 * gaussdata_eta * dux[2][2];

		for (PetscInt l=0; l<3; l++) { // fields
			for (PetscInt k=0; k<3; k++) { // directions
				dv[k][l] = w * dxdet * (dvx[0][l] * dx[0][k] + dvx[1][l] * dx[1][k] + dvx[2][l] * dx[2][k]);
			}
		}
}

static __global__ void MFStokesWrapper_A11_CUDA_kernel(PetscInt nel,PetscInt nen_u,PetscInt const *elnidx_u,PetscReal const *LA_gcoords,PetscScalar const *ufield,PetscReal const *gaussdata,PetscScalar *Yu)
{
	PetscScalar el_uxv[3]; // unifies elu, elx, elv
	PetscScalar dx[3][3]={0},du[3][3]={0},dv[3][3]={0};
    PetscScalar dxdet = 0;
    PetscInt elidx = (blockDim.x * blockIdx.x + threadIdx.x) / 32;  // one warp per element
    PetscInt id_in_warp = threadIdx.x % 32;
    PetscInt E = elnidx_u[nen_u*elidx+id_in_warp];
    PetscReal R[3],S[3],T[3];
    PetscInt c = id_in_warp % 3;
    PetscInt b = (id_in_warp % 9) / 3;
    PetscInt a = id_in_warp / 9;

    if (elidx >= nel)
      return;

	if (id_in_warp < Q2_NODES_PER_EL_3D) {

      for (PetscInt l=0; l<3; l++) {
        el_uxv[l] = LA_gcoords[3*E+l];
        R[l] = CUDA_D[3*c+l];
        S[l] = CUDA_B[3*b+l];
        T[l] = CUDA_B[3*a+l];
      }
	  TensorContract(R,S,T,el_uxv,dx[0]); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD,el_uxv,dx[0]);
      for (PetscInt l=0; l<3; l++) {
        R[l] = CUDA_B[3*c+l];
        S[l] = CUDA_D[3*b+l];
      }
	  TensorContract(R,S,T,el_uxv,dx[1]); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD,el_uxv,dx[1]);
      for (PetscInt l=0; l<3; l++) {
        S[l] = CUDA_B[3*b+l];
        T[l] = CUDA_D[3*a+l];
      }
	  TensorContract(R,S,T,el_uxv,dx[2]); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD,el_uxv,dx[2]);

	  JacobianInvert(dx,dxdet);

      for (PetscInt l=0; l<3; l++)
        el_uxv[l] = ufield[3*E+l];
	  TensorContract(R,S,T,el_uxv,du[2]); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD,el_uxv,du[2]);
      for (PetscInt l=0; l<3; l++) {
        S[l] = CUDA_D[3*b+l];
        T[l] = CUDA_B[3*a+l];
      }
	  TensorContract(R,S,T,el_uxv,du[1]); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD,el_uxv,du[1]);
      for (PetscInt l=0; l<3; l++) {
        R[l] = CUDA_D[3*c+l];
        S[l] = CUDA_B[3*b+l];
      }
	  TensorContract(R,S,T,el_uxv,du[0]); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD,el_uxv,du[0]);

	  QuadratureAction(gaussdata[elidx*NQP + id_in_warp],dx,dxdet,CUDA_w[id_in_warp],du,dv);

      for (PetscInt l=0; l<3; l++) {
        el_uxv[l] = 0;
        R[l] = CUDA_D[3*l + c];
        S[l] = CUDA_B[3*l + b];
        T[l] = CUDA_B[3*l + a];
      }
	  TensorContract(R,S,T,dv[0],el_uxv); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD_TRANSPOSE,dv[0],el_uxv);
      for (PetscInt l=0; l<3; l++) {
        R[l] = CUDA_B[3*l + c];
        S[l] = CUDA_D[3*l + b];
      }
	  TensorContract(R,S,T,dv[1],el_uxv); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD_TRANSPOSE,dv[1],el_uxv);
      for (PetscInt l=0; l<3; l++) {
        S[l] = CUDA_B[3*l + b];
        T[l] = CUDA_D[3*l + a];
      }
	  TensorContract(R,S,T,dv[2],el_uxv); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD_TRANSPOSE,dv[2],el_uxv);

      for (PetscInt l=0; l<3; l++) {
        atomicAdd_double(Yu + 3*E+l, el_uxv[l]);
      }
    }
}

static __global__ void set_zero_CUDA_kernel(PetscScalar *Yu, PetscInt localsize)
{
   for (PetscInt i = blockDim.x * blockIdx.x + threadIdx.x; i<localsize; i += blockDim.x * gridDim.x)
     Yu[i] = 0;
}

extern "C" {

#undef __FUNCT__
#define __FUNCT__ "MFA11SetUp_CUDA"
PetscErrorCode MFA11SetUp_CUDA(MatA11MF mf)
{
  PetscErrorCode ierr;
  MFA11CUDA      ctx;
  PetscReal      x1[3],w1[3],B[3][3],D[3][3],w[NQP];
  PetscInt       i,j,k;

  PetscFunctionBegin;
  if (mf->ctx) PetscFunctionReturn(0);
  ierr = PetscMalloc1(1,&ctx);CHKERRQ(ierr);
  ctx->state = 0;

  ctx->ufield     = NULL;
  ctx->LA_gcoords = NULL;
  ctx->gaussdata  = NULL;
  ctx->elnidx_u   = NULL;
  ctx->Yu         = NULL;

  ierr = PetscDTGaussQuadrature(3,-1,1,x1,w1);CHKERRQ(ierr);
  for (i=0; i<3; i++) {
    B[i][0] = .5*(PetscSqr(x1[i]) - x1[i]);
    B[i][1] = 1 - PetscSqr(x1[i]);
    B[i][2] = .5*(PetscSqr(x1[i]) + x1[i]);
    D[i][0] = x1[i] - .5;
    D[i][1] = -2*x1[i];
    D[i][2] = x1[i] + .5;
  }
  for (i=0; i<3; i++)
    for (j=0; j<3; j++)
      for (k=0; k<3; k++)
        w[(i*3+j)*3+k] = w1[i] * w1[j] * w1[k];

  ierr = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_D),D,     3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
  ierr = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_B),B,     3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
  ierr = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_w),w, 3 * 3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);

  mf->ctx = ctx;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MFA11Destroy_CUDA"
PetscErrorCode MFA11Destroy_CUDA(MatA11MF mf)
{
  PetscErrorCode ierr;
  MFA11CUDA      ctx;

  PetscFunctionBegin;
  ctx = (MFA11CUDA)mf->ctx;
  if (!ctx) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_USER,"CUDA MF-SpMV implementation should have a valid context");
  /* Free internal members */
  ierr = hipFree(ctx->ufield);CUDACHECK(ierr);
  ierr = hipFree(ctx->LA_gcoords);CUDACHECK(ierr);
  ierr = hipFree(ctx->gaussdata);CUDACHECK(ierr);
  ierr = hipFree(ctx->elnidx_u);CUDACHECK(ierr);
  ierr = hipFree(ctx->Yu);CUDACHECK(ierr);
  /* Free context */
  ierr = PetscFree(ctx);CHKERRQ(ierr);
  mf->ctx = NULL;

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MFStokesWrapper_A11_CUDA"
PetscErrorCode MFStokesWrapper_A11_CUDA(MatA11MF mf,Quadrature volQ,DM dau,PetscScalar ufield[],PetscScalar Yu[])
{
	PetscErrorCode ierr;
	DM cda;
	Vec gcoords;
	const PetscReal *LA_gcoords;
	PetscInt nel,nen_u,e,i,localsize;
	const PetscInt *elnidx_u;
	QPntVolCoefStokes *all_gausspoints;
	const QPntVolCoefStokes *cell_gausspoints;
    MFA11CUDA      cudactx = (MFA11CUDA)mf->ctx;

	PetscFunctionBegin;
	ierr = PetscLogEventBegin(MAT_MultMFA11_setup,0,0,0,0);CHKERRQ(ierr);

	/* setup for coords */
	ierr = DMGetCoordinateDM( dau, &cda);CHKERRQ(ierr);
	ierr = DMGetCoordinatesLocal( dau,&gcoords );CHKERRQ(ierr);
	ierr = VecGetArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);
    ierr = VecGetLocalSize(gcoords,&localsize);CHKERRQ(ierr);

	ierr = DMDAGetElements_pTatinQ2P1(dau,&nel,&nen_u,&elnidx_u);CHKERRQ(ierr);

	ierr = VolumeQuadratureGetAllCellData_Stokes(volQ,&all_gausspoints);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_setup,0,0,0,0);CHKERRQ(ierr);

    /* Set up CUDA data */
	ierr = PetscLogEventBegin(MAT_MultMFA11_copyto,0,0,0,0);CHKERRQ(ierr);

    if (!cudactx->elnidx_u) {
      ierr = hipMalloc(&cudactx->elnidx_u,        nel * nen_u * sizeof(PetscInt));CUDACHECK(ierr);
      ierr = hipMemcpy(cudactx->elnidx_u,elnidx_u,nel * nen_u * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);
    }

    if (!cudactx->ufield) {
      ierr = hipMalloc(&cudactx->ufield, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
    }
    /* ufield always needs to be copied */
    ierr = hipMemcpy(cudactx->ufield,ufield, localsize * sizeof(PetscScalar),hipMemcpyHostToDevice);CUDACHECK(ierr);

    if (!cudactx->LA_gcoords) {
      ierr = hipMalloc(&cudactx->LA_gcoords, localsize * sizeof(PetscReal));CUDACHECK(ierr);
    }
    
    if (!cudactx->gaussdata) {
      ierr = hipMalloc(&cudactx->gaussdata,nel * NQP * sizeof(PetscReal));CUDACHECK(ierr);
    }

    if (mf->state != cudactx->state) {
      ierr = hipMemcpy(cudactx->LA_gcoords,LA_gcoords, localsize * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

      PetscReal *gaussdata_host;
      ierr = PetscMalloc(nel * NQP * sizeof(PetscReal), &gaussdata_host);CHKERRQ(ierr);
      for (e=0; e<nel; e++) {
        ierr = VolumeQuadratureGetCellData_Stokes(volQ,all_gausspoints,e,(QPntVolCoefStokes**)&cell_gausspoints);CHKERRQ(ierr);
        for (i=0; i<NQP; i++) gaussdata_host[e*NQP + i] = cell_gausspoints[i].eta;
      }
      ierr = hipMemcpy(cudactx->gaussdata,gaussdata_host, nel * NQP * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);
      ierr = PetscFree(gaussdata_host);CHKERRQ(ierr);

      /* Save new state to avoid unnecessary subsequent copies */
      cudactx->state = mf->state;
    }

    if (!cudactx->Yu) {
      ierr = hipMalloc(&cudactx->Yu, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
    }

    ierr = hipDeviceSynchronize();CUDACHECK(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_copyto,0,0,0,0);CHKERRQ(ierr);

    /* CUDA entry point
     *  - inputs: elnidx_u, LA_gcoords, ufield, gaussdata
     *  - output: Yu
     */
	ierr = PetscLogEventBegin(MAT_MultMFA11_kernel,0,0,0,0);CHKERRQ(ierr);
    set_zero_CUDA_kernel<<<256,256>>>(cudactx->Yu, localsize);
    MFStokesWrapper_A11_CUDA_kernel<<<(nel-1)/WARPS_PER_BLOCK + 1, WARPS_PER_BLOCK*32>>>(nel,nen_u,cudactx->elnidx_u,cudactx->LA_gcoords,cudactx->ufield,cudactx->gaussdata,cudactx->Yu);
    ierr = hipDeviceSynchronize();CUDACHECK(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_kernel,0,0,0,0);CHKERRQ(ierr);

    PetscLogFlops((nel * 9) * 3*NQP*(6+6+6));           /* 9 tensor contractions per element */
    PetscLogFlops(nel*NQP*(14 + 1/* division */ + 27)); /* 1 Jacobi inversion per element */
    PetscLogFlops((nel * 9) * 3*NQP*(6+6+6));           /* 1 quadrature action per element */

    /* Read back CUDA data */
	ierr = PetscLogEventBegin(MAT_MultMFA11_copyfrom,0,0,0,0);CHKERRQ(ierr);
    ierr = hipMemcpy(Yu,cudactx->Yu,localsize * sizeof(PetscScalar),hipMemcpyDeviceToHost);CUDACHECK(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_copyfrom,0,0,0,0);CHKERRQ(ierr);

	ierr = VecRestoreArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);

	PetscFunctionReturn(0);
}

} /* extern C */
