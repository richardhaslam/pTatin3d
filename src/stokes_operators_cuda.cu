#include "hip/hip_runtime.h"
/*@ ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
 **
 **    Copyright (c) 2012
 **        Dave A. May [dave.may@erdw.ethz.ch]
 **        Institute of Geophysics
 **        ETH Zürich
 **        Sonneggstrasse 5
 **        CH-8092 Zürich
 **        Switzerland
 **
 **    project:    pTatin3d
 **    filename:   stokes_operators_tensor.c
 **
 **
 **    pTatin3d is free software: you can redistribute it and/or modify
 **    it under the terms of the GNU General Public License as published
 **    by the Free Software Foundation, either version 3 of the License,
 **    or (at your option) any later version.
 **
 **    pTatin3d is distributed in the hope that it will be useful,
 **    but WITHOUT ANY WARRANTY; without even the implied warranty of
 **    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
 **    See the GNU General Public License for more details.
 **
 **    You should have received a copy of the GNU General Public License
 **    along with pTatin3d. If not, see <http://www.gnu.org/licenses/>.
 **
 ** ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ @*/
// -*- indent-tabs-mode:t c-basic-offset:8 -*-

#include <petscfe.h>
#include <ptatin3d.h>
#include <ptatin3d_stokes.h>
#include <dmda_element_q2p1.h>
#include <stokes_operators.h>
#ifdef TATIN_HAVE_NVTX
#include "nvToolsExt.h"
#endif
#include <petsc/private/dmdaimpl.h> /* just used to quickly get local vector size and bs */

extern PetscLogEvent MAT_MultMFA11_stp;
extern PetscLogEvent MAT_MultMFA11_cto;
extern PetscLogEvent MAT_MultMFA11_ker;
extern PetscLogEvent MAT_MultMFA11_cfr;

/* Constant memory for D and B matrices */
__constant__ PetscReal CUDA_D[3*3], CUDA_B[3*3];


template< typename T >
void check(T result, char const *const func, const char *const file, int const line)
{
  if (result)
  {
    fprintf(stderr, "CUDA error at %s:%d code=%d (%s)\n",
        file, line, result, hipGetErrorString((hipError_t)result));
    hipDeviceReset();
    exit(EXIT_FAILURE);
  }
}
#define CUDACHECK(val)       check( (val), #val, __FILE__, __LINE__ )

#define WARPS_PER_BLOCK    4


/*for shuffle of double-precision point */
__device__ __inline__ double shfl_double(double x, int lane)
{
  // Split the double number into 2 32b registers.
  int lo, hi;
  asm volatile("mov.b64 {%0,%1}, %2;":"=r"(lo),"=r"(hi):"d"(x));
  // Shuffle the two 32b registers.
  lo = __shfl(lo,lane,32);
  hi = __shfl(hi,lane,32);
  // Recreate the 64b number.
  asm volatile("mov.b64 %0,{%1,%2};":"=d"(x):"r"(lo),"r"(hi));
  return x;
}

/*
 * Performs three tensor contractions: y[l,a,b,c] += T[a,k] S[b,j] R[c,i] x[l,k,j,i]
 */
static __device__ void TensorContract(PetscReal const *R,PetscReal const *S,PetscReal const *T,PetscReal const x[],PetscReal y[])
{
  PetscInt id_in_warp = threadIdx.x % 32;

  PetscInt c = id_in_warp % 3;
  PetscInt kj = id_in_warp / 3;
  PetscInt k3 = (id_in_warp / 9) * 3;
  PetscInt ji = id_in_warp % 9;

  for (PetscInt l=0; l<3; l++) {

    // u[l,k,j,c] = R[c,i] x[l,k,j,i]
    PetscReal result = 0;
    for (PetscInt i=0; i<3; i++) result += R[i] * shfl_double(x[l], kj*3+i);

    // v[l,k,b,c] = S[b,j] u[l,k,j,c]
    PetscReal result2 = 0;
    for (PetscInt j=0; j<3; j++) result2 += S[j] * shfl_double(result, (k3+j)*3+c);

    // y[l,a,b,c] = T[a,k] v[l,k,b,c]
    for (PetscInt k=0; k<3; k++) y[l] += T[k] * shfl_double(result2, k*9+ji);

  } // for l
}

static __device__ void JacobianInvert(PetscScalar dx[3][3],PetscScalar &dxdet)
{
  PetscScalar a[3][3];
  PetscScalar b0,b3,b6,idet;
  for (PetscInt j=0; j<3; j++) {
    for (PetscInt k=0; k<3; k++) {
      a[j][k] = dx[j][k];
    }
  }
  b0 =  (a[1][1]*a[2][2] - a[2][1]*a[1][2]);
  b3 = -(a[1][0]*a[2][2] - a[2][0]*a[1][2]);
  b6 =  (a[1][0]*a[2][1] - a[2][0]*a[1][1]);
  dxdet = a[0][0]*b0 + a[0][1]*b3 + a[0][2]*b6;
  idet = 1.0 / dxdet;
  dx[0][0] =  idet*b0;
  dx[0][1] = -idet*(a[0][1]*a[2][2] - a[2][1]*a[0][2]);
  dx[0][2] =  idet*(a[0][1]*a[1][2] - a[1][1]*a[0][2]);
  dx[1][0] =  idet*b3;
  dx[1][1] =  idet*(a[0][0]*a[2][2] - a[2][0]*a[0][2]);
  dx[1][2] = -idet*(a[0][0]*a[1][2] - a[1][0]*a[0][2]);
  dx[2][0] =  idet*b6;
  dx[2][1] = -idet*(a[0][0]*a[2][1] - a[2][0]*a[0][1]);
  dx[2][2] =  idet*(a[0][0]*a[1][1] - a[1][0]*a[0][1]);
}

static __device__ void QuadratureAction(PetscScalar gaussdata_eta_w_dxdet,  // gaussdata_eta * w * dxdet
    PetscScalar const dx[3][3],
    PetscScalar const du[3][3],
    PetscScalar dv[3][3])
{
  /* Symmetric gradient with respect to physical coordinates, xx, yy, zz, xy+yx, xz+zx, yz+zy */

  PetscScalar dux[3][3];
  for (PetscInt l=0; l<3; l++) { // fields
    for (PetscInt k=0; k<3; k++) { // directions
      dux[k][l] = du[0][l] * dx[k][0] + du[1][l] * dx[k][1] + du[2][l] * dx[k][2];
    }
  }

  PetscScalar dvx[3][3];
  dvx[0][0] = 2 * gaussdata_eta_w_dxdet * dux[0][0];
  dvx[0][1] =     gaussdata_eta_w_dxdet * (dux[0][1] + dux[1][0]);
  dvx[0][2] =     gaussdata_eta_w_dxdet * (dux[0][2] + dux[2][0]);
  dvx[1][0] =     dvx[0][1];
  dvx[1][1] = 2 * gaussdata_eta_w_dxdet * dux[1][1];
  dvx[1][2] =     gaussdata_eta_w_dxdet * (dux[1][2] + dux[2][1]);
  dvx[2][0] =     dvx[0][2];
  dvx[2][1] =     dvx[1][2];
  dvx[2][2] = 2 * gaussdata_eta_w_dxdet * dux[2][2];

  for (PetscInt l=0; l<3; l++) { // fields
    for (PetscInt k=0; k<3; k++) { // directions
      dv[k][l] = (dvx[0][l] * dx[0][k] + dvx[1][l] * dx[1][k] + dvx[2][l] * dx[2][k]);
    }
  }
}

static __global__ void MFStokesWrapper_A11_CUDA_kernel(PetscInt nel,PetscInt nen_u,PetscInt const *el_ids_colored,PetscInt const *elnidx_u,PetscReal const *LA_gcoords,PetscScalar const *ufield,PetscReal const *gaussdata_w,PetscScalar *Yu)
{
  PetscScalar el_x[3];
  PetscScalar el_uv[3]; // unifies elu, elv
  PetscScalar dx[3][3]={0},du[3][3]={0},dv[3][3]={0};
  PetscScalar dxdet = 0;
  PetscInt    elidx = (blockDim.x * blockIdx.x + threadIdx.x) / 32;  // one warp per colored element. elidx is here the index within the same color.
  PetscInt    id_in_warp = threadIdx.x % 32;
  PetscInt    E_times_3;
  PetscReal   R[3],S[3],T[3];
  PetscInt    c = id_in_warp % 3;
  PetscInt    b = (id_in_warp % 9) / 3;
  PetscInt    a = id_in_warp / 9;

  if (elidx >= nel)
    return;

  if (id_in_warp < Q2_NODES_PER_EL_3D) {

    elidx = el_ids_colored[elidx]; // get global element index
    E_times_3 = 3 * elnidx_u[nen_u*elidx+id_in_warp];

    for (PetscInt l=0; l<3; l++) {
      el_x[l] = LA_gcoords[E_times_3+l];
      el_uv[l] = ufield[E_times_3+l];
      R[l] = CUDA_D[3*c+l];
      S[l] = CUDA_B[3*b+l];
      T[l] = CUDA_B[3*a+l];
    }
    TensorContract(R,S,T,el_x, dx[0]); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD,el_uxv,dx[0]);
    TensorContract(R,S,T,el_uv,du[0]); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD,el_uxv,du[0]);

    for (PetscInt l=0; l<3; l++) {
      R[l] = CUDA_B[3*c+l];
      S[l] = CUDA_D[3*b+l];
    }
    TensorContract(R,S,T,el_x, dx[1]); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD,el_uxv,dx[1]);
    TensorContract(R,S,T,el_uv,du[1]); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD,el_uxv,du[1]);

    for (PetscInt l=0; l<3; l++) {
      S[l] = CUDA_B[3*b+l];
      T[l] = CUDA_D[3*a+l];
    }
    TensorContract(R,S,T,el_x, dx[2]); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD,el_uxv,dx[2]);
    TensorContract(R,S,T,el_uv,du[2]); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD,el_uxv,du[2]);

    JacobianInvert(dx,dxdet);

    QuadratureAction(gaussdata_w[elidx*NQP + id_in_warp] * dxdet,dx,du,dv);

    for (PetscInt l=0; l<3; l++) {
      el_uv[l] = 0;
      R[l] = CUDA_D[3*l + c];
      S[l] = CUDA_B[3*l + b];
      T[l] = CUDA_B[3*l + a];
    }
    TensorContract(R,S,T,dv[0],el_uv); //TensorContract(CUDA_D,CUDA_B,CUDA_B,GRAD_TRANSPOSE,dv[0],el_uxv);
    for (PetscInt l=0; l<3; l++) {
      R[l] = CUDA_B[3*l + c];
      S[l] = CUDA_D[3*l + b];
    }
    TensorContract(R,S,T,dv[1],el_uv); //TensorContract(CUDA_B,CUDA_D,CUDA_B,GRAD_TRANSPOSE,dv[1],el_uxv);
    for (PetscInt l=0; l<3; l++) {
      S[l] = CUDA_B[3*l + b];
      T[l] = CUDA_D[3*l + a];
    }
    TensorContract(R,S,T,dv[2],el_uv); //TensorContract(CUDA_B,CUDA_B,CUDA_D,GRAD_TRANSPOSE,dv[2],el_uxv);

    for (PetscInt l=0; l<3; l++) {
      Yu[E_times_3+l] += el_uv[l];   // Note: Coloring ensures that there are no races here!
    }
  }
}

static __global__ void set_zero_CUDA_kernel(PetscScalar *Yu, PetscInt localsize)
{
  for (PetscInt i = blockDim.x * blockIdx.x + threadIdx.x; i<localsize; i += blockDim.x * gridDim.x)
    Yu[i] = 0;
}

extern "C" {

PetscErrorCode MFA11SetUp_CUDA(MatA11MF mf)
{
  PetscErrorCode ierr;
  MFA11CUDA      cudactx;

  PetscFunctionBeginUser;
  if (mf->ctx) PetscFunctionReturn(0);
  ierr = PetscMalloc1(1,&cudactx);CHKERRQ(ierr);
  ierr = MFA11CUDA_SetUp(cudactx);CHKERRQ(ierr);
  mf->ctx = cudactx;
  PetscFunctionReturn(0);
}

PetscErrorCode MFA11CUDA_SetUp(MFA11CUDA cudactx)
{
    PetscErrorCode ierr;
    PetscReal      x1[3],w1[3],B[3][3],D[3][3];
    PetscInt       i;

    PetscFunctionBeginUser;
    cudactx->state = 0;

    cudactx->ufield             = NULL;
    cudactx->LA_gcoords         = NULL;
    cudactx->gaussdata_w        = NULL;
    cudactx->Yu                 = NULL;
    cudactx->element_colors     = 0;
    cudactx->elements_per_color = NULL;
    cudactx->el_ids_colored     = NULL;
    cudactx->elnidx_u           = NULL;

    ierr = PetscDTGaussQuadrature(3,-1,1,x1,w1);CHKERRQ(ierr);
    for (i=0; i<3; i++) {
      B[i][0] = .5*(PetscSqr(x1[i]) - x1[i]);
      B[i][1] = 1 - PetscSqr(x1[i]);
      B[i][2] = .5*(PetscSqr(x1[i]) + x1[i]);
      D[i][0] = x1[i] - .5;
      D[i][1] = -2*x1[i];
      D[i][2] = x1[i] + .5;
    }

    ierr = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_D),D,3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);
    ierr = hipMemcpyToSymbol(HIP_SYMBOL(CUDA_B),B,3 * 3 * sizeof(PetscReal));CUDACHECK(ierr);

    PetscFunctionReturn(0);
}

PetscErrorCode MFA11Destroy_CUDA(MatA11MF mf)
{
  PetscErrorCode ierr;
  MFA11CUDA      cudactx;

  PetscFunctionBeginUser;
  cudactx = (MFA11CUDA)mf->ctx;
  if (!cudactx) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_USER,"CUDA MF-SpMV implementation should have a valid context");
  ierr = MFA11CUDA_CleanUp(cudactx);CHKERRQ(ierr);
  ierr = PetscFree(cudactx);CHKERRQ(ierr);
  mf->ctx = NULL;
  PetscFunctionReturn(0);
}

PetscErrorCode MFA11CUDA_CleanUp(MFA11CUDA cudactx)
{
    PetscErrorCode ierr;
    PetscInt       i;

    PetscFunctionBeginUser;
    /* Free internal members */
    ierr = hipFree(cudactx->ufield);CUDACHECK(ierr);
    ierr = hipFree(cudactx->LA_gcoords);CUDACHECK(ierr);
    ierr = hipFree(cudactx->gaussdata_w);CUDACHECK(ierr);
    for (i=0; i<cudactx->element_colors; ++i) {
      ierr = hipFree(cudactx->el_ids_colored[i]);CUDACHECK(ierr);
    }
    ierr = PetscFree(cudactx->elements_per_color);CUDACHECK(ierr);
    ierr = PetscFree(cudactx->el_ids_colored);CUDACHECK(ierr);
    ierr = hipFree(cudactx->elnidx_u);CUDACHECK(ierr);
    ierr = hipFree(cudactx->Yu);CUDACHECK(ierr);

    PetscFunctionReturn(0);
}

PetscErrorCode CopyTo_A11_CUDA(MatA11MF mf,MFA11CUDA cudactx,const PetscScalar *ufield,const PetscReal *LA_gcoords,const PetscReal *gaussdata_host,PetscInt nel,PetscInt nen_u,const PetscInt *elnidx_u,PetscInt nnodes_local)
{
  PetscErrorCode ierr;
  PetscInt       i,j;
  PetscInt       localsize = NSD*nnodes_local;

  PetscFunctionBeginUser;

  if (!cudactx->elnidx_u) {
    ierr = hipMalloc(&cudactx->elnidx_u,        nel * nen_u * sizeof(PetscInt));CUDACHECK(ierr);
    ierr = hipMemcpy(cudactx->elnidx_u,elnidx_u,nel * nen_u * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Assign colors to elements such that there is no overlap in writes to Yu if elements are processed concurrently */
    PetscInt elements_colored = 0;
    PetscInt *element_color;
    PetscInt *Yu_color; // scratchpad

    ierr = PetscMalloc(nel * sizeof(PetscInt), &element_color);CHKERRQ(ierr);
    for (i=0; i<nel; ++i) element_color[i] = -1;
    ierr = PetscMalloc(nnodes_local * sizeof(PetscInt), &Yu_color);CHKERRQ(ierr);
    for (i=0; i<nnodes_local; ++i) Yu_color[i] = -1;

    cudactx->element_colors = 0;
    while (elements_colored < nel) {

      for (i=0; i<nel; ++i) {

        if (element_color[i] >= 0) continue;  /* element already has a color */

        /* Check if element can be colored: No corresponding index in Yu has current color */
        PetscInt can_be_colored = 1;
        for (j=0; j<nen_u; ++j) {
          if (Yu_color[elnidx_u[i*nen_u + j]] == cudactx->element_colors) {
            can_be_colored = 0;
            break;
          }
        }

        /* Color element if possible, update Yu indices to current color */
        if (can_be_colored) {
          element_color[i] = cudactx->element_colors;
          for (j=0; j<nen_u; ++j)
            Yu_color[elnidx_u[i*nen_u + j]] = cudactx->element_colors;

          ++elements_colored;
        }
      }

      ++cudactx->element_colors;
    }

    /* Generate CUDA arrays with coloring information */
    ierr = PetscMalloc(cudactx->element_colors * sizeof(PetscInt), &cudactx->elements_per_color);CHKERRQ(ierr);
    ierr = PetscMalloc(cudactx->element_colors * sizeof(PetscInt*), &cudactx->el_ids_colored);CHKERRQ(ierr);

    for (i=0; i<cudactx->element_colors; ++i) {
      /* count elements, collect element indices for this color and copy over to GPU: */
      cudactx->elements_per_color[i] = 0;
      for (j=0; j<nel; ++j) {
        if (element_color[j] == i) {
          Yu_color[cudactx->elements_per_color[i]] = j; /* Reusing Yu_color array here */
          cudactx->elements_per_color[i] += 1;
        }
      }

      ierr = hipMalloc(&cudactx->el_ids_colored[i],        cudactx->elements_per_color[i] * sizeof(PetscInt));CUDACHECK(ierr);
      ierr = hipMemcpy(cudactx->el_ids_colored[i],Yu_color,cudactx->elements_per_color[i] * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);
    }

    /* clean up */
    ierr = PetscFree(element_color);CHKERRQ(ierr);
    ierr = PetscFree(Yu_color);CHKERRQ(ierr);
  }

  if (!cudactx->ufield) {
    ierr = hipMalloc(&cudactx->ufield, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
  }
  /* ufield always needs to be copied */
  ierr = hipMemcpy(cudactx->ufield,ufield, localsize * sizeof(PetscScalar),hipMemcpyHostToDevice);CUDACHECK(ierr);

  if (!cudactx->LA_gcoords) {
    ierr = hipMalloc(&cudactx->LA_gcoords, localsize * sizeof(PetscReal));CUDACHECK(ierr);
  }

  if (!cudactx->gaussdata_w) {
    ierr = hipMalloc(&cudactx->gaussdata_w,nel * NQP * sizeof(PetscReal));CUDACHECK(ierr);
  }

  if (mf->state != cudactx->state) {
    ierr = hipMemcpy(cudactx->LA_gcoords,LA_gcoords, localsize * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Note that we populate and free gaussdata_host outside this function,
       since this data may have come from another rank with SubRepart */
    ierr = hipMemcpy(cudactx->gaussdata_w,gaussdata_host, nel * NQP * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Save new state to avoid unnecessary subsequent copies */
    cudactx->state = mf->state;
  }

  if (!cudactx->Yu) {
    ierr = hipMalloc(&cudactx->Yu, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
  }

  ierr = hipDeviceSynchronize();CUDACHECK(ierr);

  PetscFunctionReturn(0);
}


PetscErrorCode ProcessElements_A11_CUDA(MFA11CUDA cudactx,PetscInt nen_u,PetscInt localsize)
{
  PetscErrorCode ierr;
  PetscInt       i;

  PetscFunctionBegin;
  set_zero_CUDA_kernel<<<256,256>>>(cudactx->Yu, localsize);
  for (i=0; i<cudactx->element_colors; ++i) {
    MFStokesWrapper_A11_CUDA_kernel<<<(cudactx->elements_per_color[i]-1)/WARPS_PER_BLOCK + 1, WARPS_PER_BLOCK*32>>>(cudactx->elements_per_color[i],nen_u,cudactx->el_ids_colored[i],cudactx->elnidx_u,cudactx->LA_gcoords,cudactx->ufield,cudactx->gaussdata_w,cudactx->Yu);
  }
  // TODO remove??
  ierr = hipDeviceSynchronize();CUDACHECK(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode CopyFrom_A11_CUDA(MFA11CUDA cudactx,PetscScalar *Yu,PetscInt localsize)
{
  PetscErrorCode ierr;
#ifdef TATIN_HAVE_NVTX
  nvtxRangePushA(__FUNCTION__);
#endif

  PetscFunctionBegin;
  ierr = hipMemcpy(Yu,cudactx->Yu,localsize * sizeof(PetscScalar),hipMemcpyDeviceToHost);CUDACHECK(ierr);
#ifdef TATIN_HAVE_NVTX
  nvtxRangePop();
#endif
  PetscFunctionReturn(0);
}

/* Note that this requires Yu to be pinned/page-locked, and that you need a synchronization call later */
PetscErrorCode CopyFrom_A11_Async_CUDA(MFA11CUDA cudactx,PetscScalar *Yu,PetscInt localsize)
{
  PetscErrorCode ierr;
#ifdef TATIN_HAVE_NVTX
  nvtxRangePushA(__FUNCTION__);
#endif

  PetscFunctionBegin;
  ierr = hipMemcpyAsync(Yu,cudactx->Yu,localsize * sizeof(PetscScalar),hipMemcpyDeviceToHost);CUDACHECK(ierr);
#ifdef TATIN_HAVE_NVTX
  nvtxRangePop();
#endif
  PetscFunctionReturn(0);
}

PetscErrorCode MFStokesWrapper_A11_CUDA(MatA11MF mf,Quadrature volQ,DM dau,PetscScalar ufield[],PetscScalar Yu[])
{
  PetscErrorCode          ierr;
  DM                      cda;
  Vec                     gcoords;
  const PetscReal         *LA_gcoords;
  PetscInt                nel,nen_u,e,i,j,k,localsize;
  const PetscInt          *elnidx_u;
  QPntVolCoefStokes       *all_gausspoints;
  const QPntVolCoefStokes *cell_gausspoints;
  MFA11CUDA               cudactx = (MFA11CUDA)mf->ctx;

  PetscFunctionBegin;
  ierr = PetscLogEventBegin(MAT_MultMFA11_stp,0,0,0,0);CHKERRQ(ierr);

  /* setup for coords */
  ierr = DMGetCoordinateDM( dau, &cda);CHKERRQ(ierr);
  ierr = DMGetCoordinatesLocal( dau,&gcoords );CHKERRQ(ierr);
  ierr = VecGetArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);
  ierr = VecGetLocalSize(gcoords,&localsize);CHKERRQ(ierr);

  ierr = DMDAGetElements_pTatinQ2P1(dau,&nel,&nen_u,&elnidx_u);CHKERRQ(ierr);

  ierr = VolumeQuadratureGetAllCellData_Stokes(volQ,&all_gausspoints);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(MAT_MultMFA11_stp,0,0,0,0);CHKERRQ(ierr);

  /* Set up CUDA data */
  ierr = PetscLogEventBegin(MAT_MultMFA11_cto,0,0,0,0);CHKERRQ(ierr);
  {
    PetscReal *gaussdata_host=NULL;
    if(!cudactx->state) {
      PetscReal x1[3],w1[3],w[NQP];

      ierr = PetscDTGaussQuadrature(3,-1,1,x1,w1);CHKERRQ(ierr);
      for (i=0; i<3; i++)
        for (j=0; j<3; j++)
          for (k=0; k<3; k++)
            w[(i*3+j)*3+k] = w1[i] * w1[j] * w1[k];

      ierr = PetscMalloc(nel * NQP * sizeof(PetscReal), &gaussdata_host);CHKERRQ(ierr);
      for (e=0; e<nel; e++) {
        ierr = VolumeQuadratureGetCellData_Stokes(volQ,all_gausspoints,e,(QPntVolCoefStokes**)&cell_gausspoints);CHKERRQ(ierr);
        for (i=0; i<NQP; i++) gaussdata_host[e*NQP + i] = cell_gausspoints[i].eta * w[i];
      }

    }
    ierr = CopyTo_A11_CUDA(mf,cudactx,ufield,LA_gcoords,gaussdata_host,nel,nen_u,elnidx_u,localsize/NSD);CHKERRQ(ierr);

    if(gaussdata_host) {
      ierr = PetscFree(gaussdata_host);CHKERRQ(ierr);
    }
  }
  ierr = PetscLogEventEnd(MAT_MultMFA11_cto,0,0,0,0);CHKERRQ(ierr);

  /* CUDA entry point
   *  - inputs: elnidx_u, LA_gcoords, ufield, gaussdata_w
   *  - output: Yu
   */

    ierr = PetscLogEventBegin(MAT_MultMFA11_ker,0,0,0,0);CHKERRQ(ierr);
    ierr = ProcessElements_A11_CUDA(cudactx,nen_u,localsize);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_ker,0,0,0,0);CHKERRQ(ierr);

    PetscLogFlops((nel * 9) * 3*NQP*(6+6+6));           /* 9 tensor contractions per element */
    PetscLogFlops(nel*NQP*(14 + 1/* division */ + 27)); /* 1 Jacobi inversion per element */
    PetscLogFlops(nel*NQP*(5*9+6+6+6*9));               /* 1 quadrature action per element */

    /* Read back CUDA data */
    ierr = PetscLogEventBegin(MAT_MultMFA11_cfr,0,0,0,0);CHKERRQ(ierr);
    ierr = CopyFrom_A11_CUDA(cudactx,Yu,localsize);CHKERRQ(ierr);
    // TODO
    //ierr = CopyFrom_A11_Async_CUDA(cudactx,Yu,localsize);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_MultMFA11_cfr,0,0,0,0);CHKERRQ(ierr);

    ierr = VecRestoreArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);

    PetscFunctionReturn(0);
}

/* ======= cell iterator variant ======= */

PetscErrorCode CopyTo_A11_CUDA_celliterator(MatA11MF mf,MFA11CUDA cudactx,const PetscScalar *ufield,const PetscReal *LA_gcoords,const PetscReal *gaussdata_host,PetscInt nel,PetscInt nen_u,const PetscInt *elnidx_u,PetscInt nnodes_local,PetscInt ncells,PetscInt cell[])
{
  PetscErrorCode ierr;
  PetscInt       i,c,j;
  PetscInt       localsize = NSD*nnodes_local;
#ifdef TATIN_HAVE_NVTX
  nvtxRangePushA(__FUNCTION__);
#endif

  PetscFunctionBeginUser;

  if (!cudactx->elnidx_u) {
    ierr = hipMalloc(&cudactx->elnidx_u,        nel * nen_u * sizeof(PetscInt));CUDACHECK(ierr);
    ierr = hipMemcpy(cudactx->elnidx_u,elnidx_u,nel * nen_u * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Assign colors to elements such that there is no overlap in writes to Yu if elements are processed concurrently */
    PetscInt elements_colored = 0;
    PetscInt *element_color;
    PetscInt *Yu_color; // scratchpad

    ierr = PetscMalloc(nel * sizeof(PetscInt), &element_color);CHKERRQ(ierr);
    for (i=0; i<nel; ++i) element_color[i] = -1;
    ierr = PetscMalloc(nnodes_local * sizeof(PetscInt), &Yu_color);CHKERRQ(ierr);
    for (i=0; i<nnodes_local; ++i) Yu_color[i] = -1;

    cudactx->element_colors = 0;
    while (elements_colored < ncells) {

      for (c=0; c<ncells; ++c) {
        PetscInt i = cell[c];

        if (element_color[i] >= 0) continue;  /* element already has a color */

        /* Check if element can be colored: No corresponding index in Yu has current color */
        PetscInt can_be_colored = 1;
        for (j=0; j<nen_u; ++j) {
          if (Yu_color[elnidx_u[i*nen_u + j]] == cudactx->element_colors) {
            can_be_colored = 0;
            break;
          }
        }

        /* Color element if possible, update Yu indices to current color */
        if (can_be_colored) {
          element_color[i] = cudactx->element_colors;
          for (j=0; j<nen_u; ++j)
            Yu_color[elnidx_u[i*nen_u + j]] = cudactx->element_colors;

          ++elements_colored;
        }
      }

      ++cudactx->element_colors;
    }

    /* Generate CUDA arrays with coloring information */
    ierr = PetscMalloc(cudactx->element_colors * sizeof(PetscInt), &cudactx->elements_per_color);CHKERRQ(ierr);
    ierr = PetscMalloc(cudactx->element_colors * sizeof(PetscInt*), &cudactx->el_ids_colored);CHKERRQ(ierr);

    for (i=0; i<cudactx->element_colors; ++i) {
      /* count elements, collect element indices for this color and copy over to GPU: */
      cudactx->elements_per_color[i] = 0;
      for (j=0; j<nel; ++j) {
        if (element_color[j] == i) {
          Yu_color[cudactx->elements_per_color[i]] = j; /* Reusing Yu_color array here */
          cudactx->elements_per_color[i] += 1;
        }
      }

      ierr = hipMalloc(&cudactx->el_ids_colored[i],        cudactx->elements_per_color[i] * sizeof(PetscInt));CUDACHECK(ierr);
      ierr = hipMemcpy(cudactx->el_ids_colored[i],Yu_color,cudactx->elements_per_color[i] * sizeof(PetscInt),hipMemcpyHostToDevice);CUDACHECK(ierr);
    }

    /* clean up */
    ierr = PetscFree(element_color);CHKERRQ(ierr);
    ierr = PetscFree(Yu_color);CHKERRQ(ierr);
  }

  if (!cudactx->ufield) {
    ierr = hipMalloc(&cudactx->ufield, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
  }
  /* ufield always needs to be copied */
  // TODO make async (means ufield must be pinned)
  ierr = hipMemcpy(cudactx->ufield,ufield, localsize * sizeof(PetscScalar),hipMemcpyHostToDevice);CUDACHECK(ierr);

  if (!cudactx->LA_gcoords) {
    ierr = hipMalloc(&cudactx->LA_gcoords, localsize * sizeof(PetscReal));CUDACHECK(ierr);
  }

  if (!cudactx->gaussdata_w) {
    ierr = hipMalloc(&cudactx->gaussdata_w,nel * NQP * sizeof(PetscReal));CUDACHECK(ierr);
  }

  if (mf->state != cudactx->state) {
    ierr = hipMemcpy(cudactx->LA_gcoords,LA_gcoords, localsize * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Note that we populate and free gaussdata_host outside this function,
    since this data may have come from another rank with SubRepart */
    ierr = hipMemcpy(cudactx->gaussdata_w,gaussdata_host, nel * NQP * sizeof(PetscReal),hipMemcpyHostToDevice);CUDACHECK(ierr);

    /* Save new state to avoid unnecessary subsequent copies */
    cudactx->state = mf->state;
  }

  if (!cudactx->Yu) {
    ierr = hipMalloc(&cudactx->Yu, localsize * sizeof(PetscScalar));CUDACHECK(ierr);
  }

  // TODO remove?
  ierr = hipDeviceSynchronize();CUDACHECK(ierr);
#ifdef TATIN_HAVE_NVTX
  nvtxRangePop();
#endif

  PetscFunctionReturn(0);
}

PetscErrorCode MFStokesWrapper_A11_CUDA_celliterator(MatA11MF mf,Quadrature volQ,DM dau,PetscInt ncells,PetscInt cell[],PetscScalar ufield[],PetscScalar Yu[])
{
  PetscErrorCode          ierr;
  DM                      cda;
  Vec                     gcoords;
  const PetscReal         *LA_gcoords;
  PetscInt                nel,nen_u,e,i,j,k,localsize;
  const PetscInt          *elnidx_u;
  QPntVolCoefStokes       *all_gausspoints;
  const QPntVolCoefStokes *cell_gausspoints;
  MFA11CUDA               cudactx = (MFA11CUDA)mf->ctx;

#ifdef TATIN_HAVE_NVTX
  nvtxRangePushA(__FUNCTION__);
#endif

  PetscFunctionBegin;
  ierr = PetscLogEventBegin(MAT_MultMFA11_stp,0,0,0,0);CHKERRQ(ierr);

  /* setup for coords */
  ierr = DMGetCoordinateDM(dau,&cda);CHKERRQ(ierr);
  ierr = DMGetCoordinatesLocal(dau,&gcoords);CHKERRQ(ierr);
  ierr = VecGetArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);
  ierr = VecGetLocalSize(gcoords,&localsize);CHKERRQ(ierr);

  ierr = DMDAGetElements_pTatinQ2P1(dau,&nel,&nen_u,&elnidx_u);CHKERRQ(ierr);

  ierr = VolumeQuadratureGetAllCellData_Stokes(volQ,&all_gausspoints);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(MAT_MultMFA11_stp,0,0,0,0);CHKERRQ(ierr);

  /* Set up CUDA data */
  ierr = PetscLogEventBegin(MAT_MultMFA11_cto,0,0,0,0);CHKERRQ(ierr);
  {
    PetscReal *gaussdata_host=NULL;
    if(!cudactx->state) {
      PetscReal x1[3],w1[3],w[NQP];

      ierr = PetscDTGaussQuadrature(3,-1,1,x1,w1);CHKERRQ(ierr);
      for (i=0; i<3; i++)
        for (j=0; j<3; j++)
          for (k=0; k<3; k++)
            w[(i*3+j)*3+k] = w1[i] * w1[j] * w1[k];

      /* Note that we waste some effort still transferring all of the gaussdata, even though not
         all elements are processed. */
      ierr = PetscMalloc(nel * NQP * sizeof(PetscReal), &gaussdata_host);CHKERRQ(ierr);
      for (e=0; e<nel; e++) {
        ierr = VolumeQuadratureGetCellData_Stokes(volQ,all_gausspoints,e,(QPntVolCoefStokes**)&cell_gausspoints);CHKERRQ(ierr);
        for (i=0; i<NQP; i++) gaussdata_host[e*NQP + i] = cell_gausspoints[i].eta * w[i];
      }

      }
      ierr = CopyTo_A11_CUDA_celliterator(mf,cudactx,ufield,LA_gcoords,gaussdata_host,nel,nen_u,elnidx_u,localsize/NSD,ncells,cell);CHKERRQ(ierr);

      if(gaussdata_host) {
      ierr = PetscFree(gaussdata_host);CHKERRQ(ierr);
    }
  }
  ierr = PetscLogEventEnd(MAT_MultMFA11_cto,0,0,0,0);CHKERRQ(ierr);

  /* CUDA entry point
  *  - inputs: elnidx_u, LA_gcoords, ufield, gaussdata_w
  *  - output: Yu
  */

  ierr = PetscLogEventBegin(MAT_MultMFA11_ker,0,0,0,0);CHKERRQ(ierr);
  ierr = ProcessElements_A11_CUDA(cudactx,nen_u,localsize);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(MAT_MultMFA11_ker,0,0,0,0);CHKERRQ(ierr);

  PetscLogFlops((ncells * 9) * 3*NQP*(6+6+6));           /* 9 tensor contractions per element */
  PetscLogFlops(ncells*NQP*(14 + 1/* division */ + 27)); /* 1 Jacobi inversion per element */
  PetscLogFlops(ncells*NQP*(5*9+6+6+6*9));               /* 1 quadrature action per element */

  /* Read back CUDA data */
  ierr = PetscLogEventBegin(MAT_MultMFA11_cfr,0,0,0,0);CHKERRQ(ierr);
  ierr = CopyFrom_A11_CUDA(cudactx,Yu,localsize);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(MAT_MultMFA11_cfr,0,0,0,0);CHKERRQ(ierr);

  ierr = VecRestoreArrayRead(gcoords,&LA_gcoords);CHKERRQ(ierr);

#ifdef TATIN_HAVE_NVTX
  nvtxRangePop();
#endif

  PetscFunctionReturn(0);
}

PetscErrorCode DMDACreateLocalVectorPinnedSeq_CUDA(DM da, Vec *vec)
{
  PetscErrorCode ierr;
  DM_DA          *dd = (DM_DA*)da->data;
  PetscScalar    *array;

  PetscFunctionBeginUser;
  PetscValidHeaderSpecificType(da,DM_CLASSID,1,DMDA);
  ierr = hipHostMalloc(&array,dd->nlocal * sizeof(PetscScalar)); CUDACHECK(ierr);/* Must be freed later with hipFree() */
  ierr = VecCreateSeqWithArray(PETSC_COMM_SELF,dd->w,dd->nlocal,array,vec);CHKERRQ(ierr);
  ierr = VecSetDM(*vec,da);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode VecDestroyPinnedSeq_CUDA(Vec *vec)
{
  PetscErrorCode ierr;
  PetscScalar    *arr;

  PetscFunctionBeginUser;
  ierr = VecGetArray(*vec,&arr);CHKERRQ(ierr); /* Is it safe to do this? */
  //ierr = hipFree(arr);CUDACHECK(ierr);
  // TODO do this properly (errors if you check!)
  ierr = hipFree(arr);
  ierr = VecPlaceArray(*vec,NULL);CHKERRQ(ierr);
  ierr = VecDestroy(vec);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

PetscErrorCode Synchronize_CUDA()
{
  PetscErrorCode ierr;

  PetscFunctionBeginUser;
  ierr = hipDeviceSynchronize();CUDACHECK(ierr);
  PetscFunctionReturn(0);

}


} /* extern C */
